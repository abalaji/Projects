#include "hip/hip_runtime.h"
/*
=============================================
Author:Anusha Balaji & Lalitha Geddapu
Name:insertion_omp.c
Description: Insertion Sort in CUDA
=============================================
*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define MAX 32678

void ms(int a[],int l,int m,int h);
void part(int a[],int l,int h);
int checkError(int a[], int c[], int n);

__global__ void insert(int *a, int n)
{
int i = blockIdx.x *n;
int c,d,p,t,size;
size = (blockIdx.x+1)*n;

	for(c=i;c<size;c++)
	{
	d=c;
	while(d>i&&a[d]<a[d-1])
	{
	t=a[d];
	a[d]=a[d-1];
	a[d-1]=t;
	d--;
	}
	}

}	

		
int main()
{
int i,p,k,r;
int num_elem, num_bytes;
int *device_aay, *host_aay, *checkaay;
double res_time[45];
p=0;
FILE *insertcuda;
int block_size;

for(k=1; k<15; k++)
{	
	num_elem=pow(2,k);
	
	//computing the size in bytes
	num_bytes=num_elem * sizeof(int);

	//malloc host_aay
	host_aay=(int*)malloc(num_bytes);
	checkaay=(int*)malloc(num_bytes);

	//hipMalloc device aay
	hipMalloc((void**)&device_aay,num_bytes);

		//initialising host aay
		for (i=0;i<num_elem;i++)
		{
			host_aay[i]=rand()%num_elem;
			checkaay[i]=host_aay[i];
		}
		
		block_size=8;
		hipMemcpy(device_aay,host_aay,num_bytes,hipMemcpyHostToDevice);
		hipEvent_t start_insert, stop_insert, start_merge, stop_merge;
		hipEventCreate(&start_insert);
		hipEventCreate(&start_merge);
		hipEventCreate(&stop_insert);
		hipEventCreate(&stop_merge);
		
		hipEventRecord(start_insert,0);
		
		for (r = 0; r < 1000; r++)
		{
		insert<<<block_size,1>>>(device_aay,num_elem/block_size);
		}
		
		hipEventRecord(stop_insert,0);
		hipEventSynchronize(stop_insert);
		float elapsedTimeInsert;
		hipEventElapsedTime(&elapsedTimeInsert, start_insert,stop_insert);
		
		hipMemcpy(host_aay,device_aay,num_bytes,hipMemcpyDeviceToHost);
		
		hipEventRecord(start_merge,0);
		
		part(host_aay,0,num_elem-1);
		
		hipEventRecord(stop_merge,0);
		hipEventSynchronize(stop_merge);
		float elapsedTimeMerge;
		hipEventElapsedTime(&elapsedTimeMerge, start_merge,stop_merge);

		part(checkaay,0,num_elem-1);

	/*printf("\n\n");
	
	printf ("Time for the insertion sort: %f ms\n", elapsedTimeInsert);
	printf ("Time for the merge sort: %f ms\n", elapsedTimeMerge);
	
	printf("\n\n");*/
	
	/*missorted = checkError(host_aay,checkaay,num_elem);
    if (missorted != 0) printf("%d missorted nubmers\n",missorted);*/

	res_time[p]= num_elem;
	res_time[p+1]=elapsedTimeInsert;
	res_time[p+2]=elapsedTimeMerge;
	p=p+3;
	
	//deallocate memory
	free(host_aay);
	free(checkaay);
	hipFree(device_aay);
}

	insertcuda=fopen("insertcuda.csv","w");
	if(!insertcuda)
	{
		printf("file opening failed");
		fclose(insertcuda);
	}

	/* Calculation Of time */
	for(p=0;p<45;p=p+3)
	{
	fprintf(insertcuda,"n=%f,insert=%f,merge=%f \n ",res_time[p],res_time[p+1],res_time[p+2]);
	}
	
	fclose(insertcuda);
	
return 0;
}

void part(int a[],int l,int h){

    int m;

    if(l<h){
         m=(l+h)/2;
         part(a,l,m);
         part(a,m+1,h);
         ms(a,l,m,h);
    }
}

void ms(int a[],int l,int m,int h){

    int i,m,k,l,temp[MAX];

    l=l;
    i=l;
    m=m+1;

    while((l<=m)&&(m<=h)){

         if(a[l]<=a[m]){
             temp[i]=a[l];
             l++;
         }
         else{
             temp[i]=a[m];
             m++;
         }
         i++;
    }

    if(l>m){
         for(k=m;k<=h;k++){
             temp[i]=a[k];
             i++;
         }
    }
    else{
         for(k=l;k<=m;k++){
             temp[i]=a[k];
             i++;
         }
    }
   
    for(k=l;k<=h;k++){
         a[k]=temp[k];
    }
}

int checkError(int a[], int c[], int n) {
    int result = 0;
    for (int i=0; i<n; i++) {
        if (a[i] != c[i]) {
            result++;
        }
    }
    return result;
}
