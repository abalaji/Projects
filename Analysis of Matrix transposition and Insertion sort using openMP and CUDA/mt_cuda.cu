/*
=============================================
Author:Anusha Balaji & Lalitha Geddapu
Name:insertion_omp.c
Description: Matrix Transposition in CUDA
=============================================
*/
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define ThPBlck 16

__global__ void transpose(float* A,float* At,int rows,int cols)
{
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int r = blockIdx.y * blockDim.x + threadIdx.y;
	
	if(c < cols && r < rows)
	{
	At[c*rows+r]=A[c+r*cols];
	}
}


//Matrix transpose Host code
int main()
{
	int i,k,q,p,r,rows,cols ;
	float* A;
	float* At;	
	float* A_d;
	float* At_d;
	float elapsedTimeTrans;
	double res_time[60];
	p=0;
	FILE *mtcuda;
	
for(k=1,q=20; k<20,q>0;k++,q--)
{
	rows = pow(2,k);
	cols = pow(2,q);
	
	size_t size = rows*cols* sizeof(float);
	
	A = (float*)malloc(size);
	At = (float*)malloc(size);
	
	hipMalloc((float**)&A_d,size);
	hipMalloc((float**)&At_d,size);
	
	/*initialize matrix in host memory*/
	for( i=0; i < rows*cols; i++)
	{
		A[i] = rand() % (rows*cols);
	}
	
	/*copy matrix from Host to device memory*/
	hipMemcpy(A_d,A,size,hipMemcpyHostToDevice);
	
	/*calculating size of grid*/
	int grid_rows = (rows + ThPBlck - 1) / ThPBlck;
    int grid_cols = (cols + ThPBlck - 1) / ThPBlck;
 
    dim3 blockSize(ThPBlck, ThPBlck);
	dim3 gridSize(grid_cols, grid_rows);	
	
	/*CUDA timer declarations*/
	hipEvent_t start_transpose, stop_transpose;
	hipEventCreate(&start_transpose);
	hipEventCreate(&stop_transpose);
	
	hipEventRecord(start_transpose,0); /*start timer*/
	
	for (r = 0; r < 1000; r++)
	{
	transpose<<<gridSize,blockSize>>>(A_d,At_d,rows,cols);
	}
	
	hipEventRecord(stop_transpose,0); /*stop timer*/
	hipEventSynchronize(stop_transpose);
	hipEventElapsedTime(&elapsedTimeTrans, start_transpose,stop_transpose);
	
	/*copy output from device to host memory*/
	hipMemcpy(At,At_d, size, hipMemcpyDeviceToHost);
	
	//printf ("\n Time for transpose: %f ms \n", elapsedTimeTrans);
	
	res_time[p]= rows;
	res_time[p+1]=cols;
	res_time[p+2]=elapsedTimeTrans;
	p=p+3;
	
	free(A);
	free(At);
	hipFree(A_d);
	hipFree(At_d);
}
	
	mtcuda=fopen("mtcuda.csv","w");
	if(!mtcuda)
	{
		printf("file opening failed");
		fclose(mtcuda);
	}

	/* Calculation Of time */
	for(p=0;p<60;p=p+3)
	{
	fprintf(mtcuda,"m=%f,n=%f,%f \n ",res_time[p],res_time[p+1],res_time[p+2]);
	}
	
	fclose(mtcuda);
	
	
return 0;
}
